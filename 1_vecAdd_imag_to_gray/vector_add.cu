#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/
#include <stdio.h>
#include "support.h"
#include "kernel.cu"

void verify(float *A, float *B, float *C, int n)
{
	const float relativeTolerance = 1e-6;
	for (int i = 0; i < n; i++)
	{
		float sum = A[i] + B[i];
		float relativeError = (sum - C[i]) / sum;
		if (relativeError > relativeTolerance ||
			relativeError < -relativeTolerance)
		{
			printf("TEST FAILED\n\n");
			exit(0);
		}
	}

	printf("TEST PASSED\n\n");
}

int main(int argc, char **argv)
{
	Timer timer;
	hipError_t cuda_ret;

	// Initialize host variables ----------------------------------------------
	printf("\nSetting up the problem...");
	fflush(stdout);
	startTime(&timer);

	unsigned int n;
	if (argc == 1)
	{
		n = 10000;
	}
	else if (argc == 2)
	{
		n = atoi(argv[1]);
	}
	else
	{
		printf("\n    Invalid input parameters!"
			"\n    Usage: ./vector_add       # Vector of size 10,000 is used"
			"\n    Usage: ./vector_add <m>   # Vector of size m is used"
			"\n");
		exit(0);
	}
/*assigns the arrays values from 0 to 0.99*/
	float *A_h = (float*) malloc(sizeof(float) *n);
	for (unsigned int i = 0; i < n; i++)
	{
		A_h[i] = (rand() % 100) / 100.00;
	}

	float *B_h = (float*) malloc(sizeof(float) *n);
	for (unsigned int i = 0; i < n; i++)
	{
		B_h[i] = (rand() % 100) / 100.00;
	}

	float *C_h = (float*) malloc(sizeof(float) *n);

	stopTime(&timer);
	printf("%f s\n", elapsedTime(timer));
	printf("    Vector size = %u\n", n);

	// Allocate device variables ----------------------------------------------
	printf("Allocating device variables...");
	fflush(stdout);
	startTime(&timer);

	float *d_a, *d_b, *d_c;
	//INSERT CODE HERE 
	int size = n * sizeof(float);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	hipDeviceSynchronize();
	stopTime(&timer);

	printf("%f s\n", elapsedTime(timer));

	// Copy host variables to device ------------------------------------------
	printf("Copying data from host to device...");
	fflush(stdout);

	startTime(&timer);
	//INSERT CODE HERE
	hipMemcpy(d_a, A_h, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, B_h, size, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	stopTime(&timer);
	printf("%f s\n", elapsedTime(timer));

	// Launch kernel ----------------------------------------------------------
	printf("Launching kernel...");
	fflush(stdout);
	startTime(&timer);

	//INSERT CODE HERE
	float BlkSize = 256.0;
	vecAddKernel<<<ceil(n/BlkSize), BlkSize>>>(d_a,d_b,d_c,n);

	cuda_ret = hipDeviceSynchronize();
	if (cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
	stopTime(&timer);
	printf("%f s\n", elapsedTime(timer));

	// Copy device variables from host ----------------------------------------
	printf("Copying data from device to host...");
	fflush(stdout);
	startTime(&timer);
	//INSERT CODE HERE
	hipMemcpy(C_h, d_c, size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	stopTime(&timer);
	printf("%f s\n", elapsedTime(timer));

	// Verify correctness -----------------------------------------------------
	printf("Verifying results...");
	fflush(stdout);
	verify(A_h, B_h, C_h, n);

	// Free memory ------------------------------------------------------------
	free(A_h);
	free(B_h);
	free(C_h);
	//INSERT CODE HERE
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
